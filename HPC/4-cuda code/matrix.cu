
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 3  // Size of matrices (3x3)

// CUDA kernel for matrix multiplication
__global__ void matrixMultiply(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    // Matrices dimensions
    const int size = N * N;

    // Host matrices
    int h_a[size], h_b[size], h_c[size];

    // Input matrices from user
    printf("Enter elements of matrix A (3x3):\n");
    for (int i = 0; i < size; ++i) {
        scanf("%d", &h_a[i]);
    }

    printf("Enter elements of matrix B (3x3):\n");
    for (int i = 0; i < size; ++i) {
        scanf("%d", &h_b[i]);
    }

    // Allocate memory for matrices on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copy matrices from host to device
    hipMemcpy(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimBlock(3, 3);
    dim3 dimGrid(1, 1);

    // Launch kernel
    matrixMultiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);

    // Copy result matrix from device to host
    hipMemcpy(h_c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Display result
    printf("Result Matrix:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d\t", h_c[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
